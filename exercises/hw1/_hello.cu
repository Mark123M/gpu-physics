#include "hip/hip_runtime.h"
#include "_hello.cuh"

__global__ void hello() {

	printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

void helloWorld() {
	hello << <2, 2 >> > ();
	hipDeviceSynchronize();
}

