#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

void helloWorld() {
	hello<<<2, 2>>>();
	hipDeviceSynchronize();
}

