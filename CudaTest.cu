// GPU function called from host code or device code
/*__global__ void mykernel(void) {

}

// Call to device code with kernel launch config
mykernel<<<1, 1>>>();

// Device pointers, just numbers
// Device pointers shouldn't be dereferenced in host code
// Host pointers shouldn't be dereferenced in device code
cudaMalloc(), cudaFree(), cudaMemcpy()

// Launch N copies of kernel to work on N instances
add<<<N, 1>>>(); // N is length of vector

blockIdx.xyz // Represents block/worker (parallel invocation of kernel) structure in multiple dimensions
*/
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <iostream>

__global__ void add(int* a, int* b, int* c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512
int main(void) {
	int* a, * b, * c; // host copies of a, b, c
	int* d_a, * d_b, * d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size); // returns error code if something goes wrong
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU with N blocks/workers
	add<<<N,1>>>(d_a, d_b, d_c);
	// Copy results back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// there could be template generic kernels :O
	for (int i = 0; i < N; i++) {
		std::cout << c[i];
	}
	std::cout << std::endl;

	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;

}